#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void cuda_hello() {
	printf("Hello World from GPU!\n");
}

int main() {
	int		deviceCount;
	hipDeviceProp_t	devProp;

	hipGetDeviceCount(&deviceCount);
	printf("Found %d devices\n", deviceCount);

	for (int device = 0; device < deviceCount; device++)
	{
		hipGetDeviceProperties(&devProp, device);
		printf("Device %d\n", device);
		printf("Compute capability     : %d.%d\n", devProp.major, devProp.minor);
		printf("Name                   : %s\n", devProp.name);
		printf("Total Global Memory    : %u\n", devProp.totalGlobalMem);
		printf("Shared memory per block: %d\n", devProp.sharedMemPerBlock);
		printf("Registers per block    : %d\n", devProp.regsPerBlock);
		printf("Warp size              : %d\n", devProp.warpSize);
		printf("Max threads per block  : %d\n", devProp.maxThreadsPerBlock);
		printf("Total constant memory  : %d\n", devProp.totalConstMem);
		printf("maxBlocksPerMultiProcessor  : %d\n", devProp.maxBlocksPerMultiProcessor);
		printf("maxThreadsPerMultiProcessor  : %d\n", devProp.maxThreadsPerMultiProcessor);
		printf("Multiprocessor count  : %d\n", devProp.multiProcessorCount);

	}

	hipEvent_t start, stop;		//описываем переменные типа  hipEvent_t 
	float       gpuTime = 0.0f;
	// создаем события начала и окончания выполнения ядра 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//привязываем событие start  к данному месту 
	hipEventRecord(start, 0);

	// вызвать ядро 
	cuda_hello << <1, 1 >> > ();

	//привязываем событие stop  к данному месту 
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	// запрашиваем время между событиями 
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("time spent executing by the GPU: %.5f ms\n", gpuTime);
	// уничтожаем созданные события 
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
