#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define VAR  0;

void checkError(hipError_t cudaStatus, char* msg, int* dev_a, int* dev_b, int* dev_c);

hipError_t multiplyWithCuda(int* c, const int* a, const int* b, unsigned int size);

void printMatrix(int* matrix, int size);

void printArray(int* a, int size);



// вариант 1
#if VAR == 1

__global__ void multiplyKernel(int* c, const int* a, const int* b)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] * b[i];
}

#else

__global__ void multiplyKernel(int* c, int* a, int* b, unsigned int size)
{
	int index = blockIdx.y * blockDim.x * blockDim.y * gridDim.x + blockIdx.x * blockDim.y + threadIdx.y * blockDim.x * gridDim.x + threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	index -= (blockDim.x * gridDim.x - size) * j;
	if (i >= size || j >= size)
		return;

	for (size_t k = 0; k < size; k++)
	{
		c[index] += a[j * size + k] * b[k * size + i];
	}
}

#endif

int main()
{
	// вариант 1
#if VAR == 1
	printf("Variant 1\n");
	printf("Enter array size: ");
	int arraySize;
	scanf("%d", &arraySize);
	int* a = (int*)malloc(arraySize * sizeof(int));
	int* b = (int*)malloc(arraySize * sizeof(int));
	int* c = (int*)malloc(arraySize * sizeof(int));
	for (size_t i = 0; i < arraySize; i++)
	{
		a[i] = rand() % 100 + 1;
		b[i] = rand() % 100 + 1;
		c[i] = 0;
	}

#else

	// вариант 0
	printf("Variant 0\n");
	int arraySize;
	scanf("%d", &arraySize);
	int* a = (int*)malloc(arraySize * arraySize * sizeof(int));
	int* b = (int*)malloc(arraySize * arraySize * sizeof(int));
	int* c = (int*)malloc(arraySize * arraySize * sizeof(int));

	for (size_t i = 0; i < arraySize; i++)
	{
		for (size_t j = 0; j < arraySize; j++)
		{
			a[i * arraySize + j] = rand() % 100 + 1;
			b[i * arraySize + j] = rand() % 100 + 1;
			c[i * arraySize + j] = 0;
		}
	}

#endif

	// Add vectors in parallel.
	hipError_t cudaStatus = multiplyWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

#if VAR == 1

	printf("A: ");
	printArray(a, arraySize);

	printf("\nB: ");
	printArray(b, arraySize);

	printf("\nC: ");
	int sum = 0;
	for (int i = 0; i < arraySize; i++) {
		sum += c[i];
	}
	printf("%d", sum);

#else

	/*printf("A:\n");
	printMatrix(a, arraySize);
	printf("B:\n");
	printMatrix(b, arraySize);
	printf("C:\n");
	printMatrix(c, arraySize);*/
	for (int i = 0; i < arraySize; i++)
		for (int j = 0; j < arraySize; j++)
		{
			int test = 0;
			for (int k = 0; k < arraySize; k++)
				test += a[i * arraySize + k] * b[k * arraySize + j];
			if (test != c[i * arraySize + j]) {
				printf("wrong answer");
			}
		}

#endif

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t multiplyWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
	hipDeviceProp_t	devProp;
	hipGetDeviceProperties(&devProp, 0);
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

#if VAR == 1
	int dev_size = size;
#else
	int dev_size = size * size;
#endif

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	checkError(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", dev_a, dev_b, dev_c);

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, dev_size * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	cudaStatus = hipMalloc((void**)&dev_a, dev_size * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	cudaStatus = hipMalloc((void**)&dev_b, dev_size * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, dev_size * sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	cudaStatus = hipMemcpy(dev_b, b, dev_size * sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	cudaStatus = hipMemcpy(dev_c, c, dev_size * sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	// Launch a kernel on the GPU with one thread for each element.
	hipEvent_t start, stop;		//описываем переменные типа  hipEvent_t 
	float       gpuTime = 0.0f;
	// создаем события начала и окончания выполнения ядра 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//привязываем событие start  к данному месту 
	hipEventRecord(start, 0);
#if VAR == 1
	
	dim3 threads;
	if (dev_size < devProp.maxThreadsPerBlock) {
		threads = dim3(dev_size, 1, 1);
	}
	else {
		threads = dim3(devProp.maxThreadsPerBlock, 1, 1);
	}
	dim3 blocks(ceil(dev_size / threads.x), 1, 1);
	multiplyKernel << <blocks, threads >> > (dev_c, dev_a, dev_b);
#else
	double d;
	if (size < sqrt(devProp.maxThreadsPerBlock)) {
		d = size;
	}
	else {
		d = sqrt(devProp.maxThreadsPerBlock);
	}
	d = 16;
	dim3 threads(d, d, 1);
	dim3 blocks(ceil(size / d), ceil(size / d), 1);
	multiplyKernel << <blocks, threads >> > (dev_c, dev_a, dev_b, size);

#endif

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	checkError(cudaStatus, "multiplyKernel launch failed: %s\n", dev_a, dev_b, dev_c);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	checkError(cudaStatus, "hipDeviceSynchronize returned error code %d after launching multiplyKernel!\n", dev_a, dev_b, dev_c);

	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	// запрашиваем время между событиями 
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("time spent executing by the GPU: %.5f ms\n", gpuTime);
	// уничтожаем созданные события 
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, dev_size * sizeof(int), hipMemcpyDeviceToHost);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	return cudaStatus;
}


void checkError(hipError_t cudaStatus, char* msg, int* dev_a, int* dev_b, int* dev_c)
{
	if (cudaStatus == hipSuccess)
		return;

	printf(msg);
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}

void printArray(int* a, int size)
{
	for (int i = 0; i < size; i++)
		printf("%d ", a[i]);
}

void printMatrix(int* matrix, int size)
{
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
			printf("%d ", matrix[i * size + j]);
		printf("\n");
	}
	printf("\n");
}