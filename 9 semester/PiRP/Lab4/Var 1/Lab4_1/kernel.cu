#include "hip/hip_runtime.h"
﻿// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#define _USE_MATH_DEFINES
#include <math.h>

#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>  // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <hip/hip_vector_types.h>

// CUDA helper functions
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error check

#define MAX_EPSILON_ERROR 5e-3f

// Define the files that are to be save and the reference images for validation
//const char *imageFilename = "teapot512.pgm";
//const char* imageFilename = "dwsample-ppm-1920.ppm";
const char* imageFilename = "teapot1024.ppm";

const char* sampleName = "simpleTexture";

////////////////////////////////////////////////////////////////////////////////
// Constants
int iterations = 1;
double filter_sigma = 0.84089642; // при увеличении сигма усиливается блюр, но надо повышать яркость пикселей, можно умножать на константу в d_gaussianfilter_rgba_x (line 200) константу подбирать надо вручную (наверное существуют методы нормализации)
//double filter_sigma = 10.0;
int gaussian_radius = 3; // радиус больше делать нет смысла
int nthreads = 64;
unsigned int width, height;
unsigned int* h_img = NULL;
unsigned int* d_img = NULL;
unsigned int* d_temp = NULL;
double* gaussian_matrix = NULL;
hipArray* d_array, * d_tempArray;
hipTextureObject_t tex;
hipTextureObject_t texTempArray;
hipTextureObject_t rgbaTex;
hipTextureObject_t rgbaTexTempArray;

void loadImageData(int argc, char** argv) {
    // load image (needed so we can get the width and height before we create the
    // window
    char* image_path = NULL;

    if (argc >= 1) {
        image_path = sdkFindFilePath(imageFilename, argv[0]);
    }

    if (image_path == 0) {
        printf("Error finding image file '%s'\n", imageFilename);
        exit(EXIT_FAILURE);
    }

    sdkLoadPPM4(image_path, (unsigned char**)&h_img, &width, &height);

    if (!h_img) {
        printf("Error opening file '%s'\n", image_path);
        exit(EXIT_FAILURE);
    }

    printf("Loaded '%s', %d x %d pixels\n", image_path, width, height);
}

extern "C" void initTexture(int width, int height, void* pImage) {
    // copy image data to array
    hipChannelFormatDesc channelDesc;
    channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    checkCudaErrors(hipMallocArray(&d_array, &channelDesc, width, height));

    size_t bytesPerElem = sizeof(uchar4);
    checkCudaErrors(hipMemcpy2DToArray(
        d_array, 0, 0, pImage, width * bytesPerElem, width * bytesPerElem, height,
        hipMemcpyHostToDevice));

    checkCudaErrors(hipMallocArray(&d_tempArray, &channelDesc, width, height));

    // set texture parameters
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_array;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(hipCreateTextureObject(&rgbaTex, &texRes, &texDescr, NULL));

    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_tempArray;

    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(
        hipCreateTextureObject(&rgbaTexTempArray, &texRes, &texDescr, NULL));

    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_array;

    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModePoint;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));

    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_tempArray;

    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModePoint;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(
        hipCreateTextureObject(&texTempArray, &texRes, &texDescr, NULL));
}

extern "C" void freeTextures() {
    checkCudaErrors(hipDestroyTextureObject(tex));
    checkCudaErrors(hipDestroyTextureObject(texTempArray));
    checkCudaErrors(hipDestroyTextureObject(rgbaTex));
    checkCudaErrors(hipDestroyTextureObject(rgbaTexTempArray));
    checkCudaErrors(hipFreeArray(d_array));
    checkCudaErrors(hipFreeArray(d_tempArray));
}

// RGBA version
// reads from 32-bit unsigned int array holding 8-bit RGBA

// convert floating point rgba color to 32-bit integer
__device__ unsigned int rgbaFloatToInt(float4 rgba) {
    rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return ((unsigned int)(rgba.w * 255.0f) << 24) |
        ((unsigned int)(rgba.z * 255.0f) << 16) |
        ((unsigned int)(rgba.y * 255.0f) << 8) |
        ((unsigned int)(rgba.x * 255.0f));
}

__device__ float4 rgbaIntToFloat(unsigned int c) {
    float4 rgba;
    rgba.x = (c & 0xff) * 0.003921568627f;          //  /255.0f;
    rgba.y = ((c >> 8) & 0xff) * 0.003921568627f;   //  /255.0f;
    rgba.z = ((c >> 16) & 0xff) * 0.003921568627f;  //  /255.0f;
    rgba.w = ((c >> 24) & 0xff) * 0.003921568627f;  //  /255.0f;
    return rgba;
}


// row pass using texture lookups
__global__ void d_gaussianfilter_rgba_x(unsigned int* od, int w, int h, int r, double* gaussian_matrix, hipTextureObject_t rgbaTex) {
    unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;
    int gaussian_m_w = (r << 1) + 1;

    // as long as address is always less than height, we do work
    if (y < h) {

        for (int x = 0; x < w; x++) {
            float4 t = make_float4(0.0f);
            for (int xx = -r; xx <= r; xx++) {
                for (int yy = -r; yy <= r; yy++) {
                    t += tex2D<float4>(rgbaTex, x + xx, y + yy) * gaussian_matrix[(xx + r) * gaussian_m_w + (yy + r)];
                }
            }

            od[y * w + x] = rgbaFloatToInt(t);
        }
    }
}

__global__ void init_gaussian_matrix(double sigma, int r, double* gaussianMatrix) {
    int x = threadIdx.x;
    int y = threadIdx.y;
    int g_x = x - r;
    int g_y = y - r;
    int g_x2 = pow(g_x, 2);
    int g_y2 = pow(g_y, 2);
    double s2 = pow(sigma, 2);
    int width = (r << 1) + 1;

    gaussianMatrix[x * width + y] = 1.0 / (2.0 * M_PI * s2) * (1.0 / pow(M_E, (double)(g_x2 + g_y2) / (2.0 * s2)));
}

extern "C" double gaussianFilterRGBA(unsigned int* d_src,
    unsigned int* d_dest, int width, int height,
    double sigma, int nthreads, int r, double* gaussian_matrix) {
    // var for kernel computation timing
    double dKernelTime;
    int gaussian_m_w = (r << 1) + 1;

    // sync host and start kernel computation timer_kernel
    dKernelTime = 0.0;
    checkCudaErrors(hipDeviceSynchronize());

    init_gaussian_matrix << <1, dim3(gaussian_m_w, gaussian_m_w) >> > (sigma, r, gaussian_matrix);
    checkCudaErrors(hipDeviceSynchronize());

    // use texture for horizontal pass
    d_gaussianfilter_rgba_x << <height / nthreads, nthreads, 0 >> > (d_dest, width, height, r, gaussian_matrix, rgbaTex);


    // sync host and stop computation timer_kernel
    checkCudaErrors(hipDeviceSynchronize());

    return ((dKernelTime / 1000.) / (double)iterations);
}
////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest();

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {
    char* ref_file = NULL;
    printf("%s starting...\n", sampleName);

    // Process command-line arguments
    if (argc > 1) {
        if (checkCmdLineFlag(argc, (const char**)argv, "threads")) {
            nthreads = getCmdLineArgumentInt(argc, (const char**)argv, "threads");
        }

        if (checkCmdLineFlag(argc, (const char**)argv, "radius")) {
            filter_sigma =
                getCmdLineArgumentInt(argc, (const char**)argv, "radius");
        }

        if (checkCmdLineFlag(argc, (const char**)argv, "passes")) {
            iterations = getCmdLineArgumentInt(argc, (const char**)argv, "passes");
        }

        if (checkCmdLineFlag(argc, (const char**)argv, "file")) {
            getCmdLineArgumentString(argc, (const char**)argv, "file",
                (char**)&ref_file);
        }
    }

    loadImageData(argc, argv);

    runTest();
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest() {
    char dump_file[256];
    int gaussian_m_w = (gaussian_radius < 1) + 1;
    checkCudaErrors(hipMalloc((void**)&d_img, (width * height * sizeof(unsigned int))));
    checkCudaErrors(hipMalloc((void**)&d_temp, (width * height * sizeof(unsigned int))));
    checkCudaErrors(hipMalloc((void**)&gaussian_matrix, (gaussian_m_w * gaussian_m_w * sizeof(double))));

    initTexture(width, height, h_img);

    unsigned int* d_result;
    unsigned int* h_result = (unsigned int*)malloc(width * height * sizeof(unsigned int));
    checkCudaErrors(hipMalloc((void**)&d_result, width * height * sizeof(unsigned int)));

    // run the sample radius
    {
        gaussianFilterRGBA(d_img, d_result, width, height, filter_sigma, nthreads, gaussian_radius, gaussian_matrix);

        // check if kernel execution generated an error
        getLastCudaError("Error: boxFilterRGBA Kernel execution FAILED");
        checkCudaErrors(hipDeviceSynchronize());

        // readback the results to system memory
        hipMemcpy((unsigned char*)h_result, (unsigned char*)d_result,
            width * height * sizeof(unsigned int), hipMemcpyDeviceToHost);

        sprintf(dump_file, "out_gaussian_%d.ppm", gaussian_radius);

        sdkSavePPM4ub((const char*)dump_file, (unsigned char*)h_result, width,
            height);

        printf("Image is convert");
    }
    printf("\n");

    free(h_result);
    checkCudaErrors(hipFree(d_result));
}
