#include "hip/hip_runtime.h"
﻿  // Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>  // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <hip/hip_vector_types.h>

// CUDA helper functions
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error check

#define MAX_EPSILON_ERROR 5e-3f

// Define the files that are to be save and the reference images for validation
//const char *imageFilename = "teapot512.pgm";
const char* imageFilename = "teapot1024.ppm";

const char* sampleName = "simpleTexture";

////////////////////////////////////////////////////////////////////////////////
// Constants
int iterations = 1;
int filter_radius = 3;
int nthreads = 64;
unsigned int width, height;
unsigned int* h_img = NULL;
unsigned int* d_img = NULL;
unsigned int* d_temp = NULL;
hipArray* d_array, * d_tempArray;
hipTextureObject_t tex;
hipTextureObject_t texTempArray;
hipTextureObject_t rgbaTex;
hipTextureObject_t rgbaTexTempArray;

void loadImageData(int argc, char** argv) {
    // load image (needed so we can get the width and height before we create the
    // window
    char* image_path = NULL;

    if (argc >= 1) {
        image_path = sdkFindFilePath(imageFilename, argv[0]);
    }

    if (image_path == 0) {
        printf("Error finding image file '%s'\n", imageFilename);
        exit(EXIT_FAILURE);
    }

    sdkLoadPPM4(image_path, (unsigned char**)&h_img, &width, &height);

    if (!h_img) {
        printf("Error opening file '%s'\n", image_path);
        exit(EXIT_FAILURE);
    }

    printf("Loaded '%s', %d x %d pixels\n", image_path, width, height);
}

extern "C" void initTexture(int width, int height, void* pImage) {
    // copy image data to array
    hipChannelFormatDesc channelDesc;
    channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    checkCudaErrors(hipMallocArray(&d_array, &channelDesc, width, height));

    size_t bytesPerElem = sizeof(uchar4);
    checkCudaErrors(hipMemcpy2DToArray(
        d_array, 0, 0, pImage, width * bytesPerElem, width * bytesPerElem, height,
        hipMemcpyHostToDevice));

    checkCudaErrors(hipMallocArray(&d_tempArray, &channelDesc, width, height));

    // set texture parameters
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_array;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(hipCreateTextureObject(&rgbaTex, &texRes, &texDescr, NULL));

    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_tempArray;

    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(
        hipCreateTextureObject(&rgbaTexTempArray, &texRes, &texDescr, NULL));

    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_array;

    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModePoint;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));

    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_tempArray;

    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModePoint;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(
        hipCreateTextureObject(&texTempArray, &texRes, &texDescr, NULL));
}

extern "C" void freeTextures() {
    checkCudaErrors(hipDestroyTextureObject(tex));
    checkCudaErrors(hipDestroyTextureObject(texTempArray));
    checkCudaErrors(hipDestroyTextureObject(rgbaTex));
    checkCudaErrors(hipDestroyTextureObject(rgbaTexTempArray));
    checkCudaErrors(hipFreeArray(d_array));
    checkCudaErrors(hipFreeArray(d_tempArray));
}

// RGBA version
// reads from 32-bit unsigned int array holding 8-bit RGBA

// convert floating point rgba color to 32-bit integer
__device__ unsigned int rgbaFloatToInt(float4 rgba) {
    rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return ((unsigned int)(rgba.w * 255.0f) << 24) |
        ((unsigned int)(rgba.z * 255.0f) << 16) |
        ((unsigned int)(rgba.y * 255.0f) << 8) |
        ((unsigned int)(rgba.x * 255.0f));
}

__device__ float4 rgbaIntToFloat(unsigned int c) {
    float4 rgba;
    rgba.x = (c & 0xff) * 0.003921568627f;          //  /255.0f;
    rgba.y = ((c >> 8) & 0xff) * 0.003921568627f;   //  /255.0f;
    rgba.z = ((c >> 16) & 0xff) * 0.003921568627f;  //  /255.0f;
    rgba.w = ((c >> 24) & 0xff) * 0.003921568627f;  //  /255.0f;
    return rgba;
}

// row pass using texture lookups
__global__ void d_boxfilter_rgba_x(unsigned int* od, int w, int h, int r, hipTextureObject_t rgbaTex) {
    float scale = 1.0f / (float)((r << 1) + 1);
    unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;

    // as long as address is always less than height, we do work
    if (y < h) {
        float4 t = make_float4(0.0f);

        for (int x = -r; x <= r; x++) {
            t += tex2D<float4>(rgbaTex, x, y);
        }

        od[y * w] = rgbaFloatToInt(t * scale);

        for (int x = 1; x < w; x++) {
            t += tex2D<float4>(rgbaTex, x + r, y);
            t -= tex2D<float4>(rgbaTex, x - r - 1, y);
            od[y * w + x] = rgbaFloatToInt(t * scale);
        }
    }
}

// column pass using coalesced global memory reads
__global__ void d_boxfilter_rgba_y(unsigned int* id, unsigned int* od, int w,
    int h, int r) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    id = &id[x];
    od = &od[x];

    float scale = 1.0f / (float)((r << 1) + 1);

    float4 t;
    // do left edge
    t = rgbaIntToFloat(id[0]) * r;

    for (int y = 0; y < (r + 1); y++) {
        t += rgbaIntToFloat(id[y * w]);
    }

    od[0] = rgbaFloatToInt(t * scale);

    for (int y = 1; y < (r + 1); y++) {
        t += rgbaIntToFloat(id[(y + r) * w]);
        t -= rgbaIntToFloat(id[0]);
        od[y * w] = rgbaFloatToInt(t * scale);
    }

    // main loop
    for (int y = (r + 1); y < (h - r); y++) {
        t += rgbaIntToFloat(id[(y + r) * w]);
        t -= rgbaIntToFloat(id[((y - r) * w) - w]);
        od[y * w] = rgbaFloatToInt(t * scale);
    }

    // do right edge
    for (int y = h - r; y < h; y++) {
        t += rgbaIntToFloat(id[(h - 1) * w]);
        t -= rgbaIntToFloat(id[((y - r) * w) - w]);
        od[y * w] = rgbaFloatToInt(t * scale);
    }
}

extern "C" double boxFilterRGBA(unsigned int* d_src, unsigned int* d_temp,
    unsigned int* d_dest, int width, int height,
    int radius, int iterations, int nthreads) {
    // var for kernel computation timing
    double dKernelTime;

    // sync host and start kernel computation timer_kernel
    dKernelTime = 0.0;
    checkCudaErrors(hipDeviceSynchronize());

    // use texture for horizontal pass
    d_boxfilter_rgba_x << <height / nthreads, nthreads, 0 >> > (d_temp, width, height, radius, rgbaTex);

    d_boxfilter_rgba_y << <width / nthreads, nthreads, 0 >> > (d_temp, d_dest, width, height, radius);

    // sync host and stop computation timer_kernel
    checkCudaErrors(hipDeviceSynchronize());

    if (iterations > 1) {
        // copy result back from global memory to array
        checkCudaErrors(hipMemcpy2DToArray(
            d_tempArray, 0, 0, d_dest, width * sizeof(unsigned int),
            width * sizeof(unsigned int), height, hipMemcpyDeviceToDevice));
    }

    return ((dKernelTime / 1000.) / (double)iterations);
}
////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest();

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {
    char* ref_file = NULL;
    printf("%s starting...\n", sampleName);

    // Process command-line arguments
    if (argc > 1) {
        if (checkCmdLineFlag(argc, (const char**)argv, "threads")) {
            nthreads = getCmdLineArgumentInt(argc, (const char**)argv, "threads");
        }

        if (checkCmdLineFlag(argc, (const char**)argv, "radius")) {
            filter_radius =
                getCmdLineArgumentInt(argc, (const char**)argv, "radius");
        }

        if (checkCmdLineFlag(argc, (const char**)argv, "passes")) {
            iterations = getCmdLineArgumentInt(argc, (const char**)argv, "passes");
        }

        if (checkCmdLineFlag(argc, (const char**)argv, "file")) {
            getCmdLineArgumentString(argc, (const char**)argv, "file",
                (char**)&ref_file);
        }
    }

    loadImageData(argc, argv);

    runTest();
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest() {
    char dump_file[256];

    checkCudaErrors(
        hipMalloc((void**)&d_img, (width * height * sizeof(unsigned int))));
    checkCudaErrors(
        hipMalloc((void**)&d_temp, (width * height * sizeof(unsigned int))));

    initTexture(width, height, h_img);

    unsigned int* d_result;
    unsigned int* h_result =
        (unsigned int*)malloc(width * height * sizeof(unsigned int));
    checkCudaErrors(
        hipMalloc((void**)&d_result, width * height * sizeof(unsigned int)));

    // run the sample radius
    {
        boxFilterRGBA(d_img, d_temp, d_result, width, height, filter_radius,
            iterations, nthreads);

        // check if kernel execution generated an error
        getLastCudaError("Error: boxFilterRGBA Kernel execution FAILED");
        checkCudaErrors(hipDeviceSynchronize());

        // readback the results to system memory
        hipMemcpy((unsigned char*)h_result, (unsigned char*)d_result,
            width * height * sizeof(unsigned int), hipMemcpyDeviceToHost);

        sprintf(dump_file, "teapot1024_%02d.ppm", filter_radius);

        sdkSavePPM4ub((const char*)dump_file, (unsigned char*)h_result, width,
            height);

        printf("Image is convert");
    }
    printf("\n");

    free(h_result);
    checkCudaErrors(hipFree(d_result));
}
