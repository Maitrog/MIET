#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <crt/math_functions.hpp>


void printArray(int* a, int size);

void checkError(hipError_t cudaStatus, char* msg, int* dev_a, int* dev_c);

void validate(int result, int* array, int arraySize);

hipError_t reductionMinWithCuda(int* c, const int* a, unsigned int size, int threadSize, int blockSize, void(*kernel) (dim3, dim3, int, int*, int*, int));

__global__ void reductionMin4(int* inData, int* outData, int arraySize)
{
	extern __shared__ int data[];
	int tid = threadIdx.x;
	int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;
	if (i + blockDim.x < arraySize) {
		data[tid] = min(inData[i], inData[i + blockDim.x]);
	}
	else {
		data[tid] = inData[i];
	}
	__syncthreads();
	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			int v = min(data[tid], data[tid + s]);
			data[tid] = v;
		}
		__syncthreads();
	}
	if (tid == 0)
		outData[blockIdx.x] = data[0];
}

__global__ void reductionMin1(int* inData, int* outData, int arraySize)
{
	extern __shared__ int data[];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= arraySize)
		data[i] = INT_MAX;
	else
		data[tid] = inData[i]; 	// load into shared memory 
	__syncthreads();

	for (int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2 * s) == 0) 	// heavy branching !!! 
			data[tid] = min(data[tid], data[tid + s]);
		__syncthreads();
	}
	if (tid == 0)
		outData[blockIdx.x] = data[0];
}


__global__ void reductionMin2(int* inData, int* outData, int arraySize)
{
	extern __shared__ int data[];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= arraySize)
		return;

	data[tid] = inData[i]; 	// load into shared memory 
	__syncthreads();
	for (int s = 1; s < blockDim.x; s <<= 1)
	{
		int index = 2 * s * tid;
		if (index < blockDim.x)
			data[index] = min(data[index], data[index + s]);
		__syncthreads();
	}
	if (tid == 0) 		// write result of block reduction 
		outData[blockIdx.x] = data[0];
}

__global__ void reductionMin3(int* inData, int* outData, int arraySize)
{
	extern __shared__ int data[];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= arraySize)
		return;

	data[tid] = inData[i];
	__syncthreads();
	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
			data[tid] = min(data[tid], data[tid + s]);
		__syncthreads();
	}
	if (tid == 0)
		outData[blockIdx.x] = data[0];
}

void reductionMin1withCuda(dim3 blocks, dim3 threads, int threadSize, int* dev_a, int* dev_c, int size) {
	reductionMin1<<<blocks, threads, threadSize * sizeof(int)>>>(dev_a, dev_c, size);
}

void reductionMin2withCuda(dim3 blocks, dim3 threads, int threadSize, int* dev_a, int* dev_c, int size) {
	reductionMin2 << <blocks, threads, threadSize * sizeof(int) >> > (dev_a, dev_c, size);
}

void reductionMin3withCuda(dim3 blocks, dim3 threads, int threadSize, int* dev_a, int* dev_c, int size) {
	reductionMin3 << <blocks, threads, threadSize * sizeof(int) >> > (dev_a, dev_c, size);
}

void reductionMin4withCuda(dim3 blocks, dim3 threads, int threadSize, int* dev_a, int* dev_c, int size) {
	reductionMin4 << <blocks, threads, threadSize * sizeof(int) >> > (dev_a, dev_c, size);
}

int main()
{
	printf("Enter array size: ");
	int arraySize;
	scanf("%d", &arraySize);
	int threadSize = 256;
	int blockSize = ceil(arraySize / threadSize);
	int blockSize2 = ceil(arraySize / threadSize / 2);
	int* a = (int*)malloc(arraySize * sizeof(int));
	int* c = (int*)malloc(blockSize * sizeof(int));
	for (size_t i = 0; i < arraySize; i++)
	{
		a[i] = rand() * (rand() % 2 == 0 ? -1 : 1);
	}

	hipError_t cudaStatus = reductionMinWithCuda(c, a, arraySize, threadSize, blockSize, reductionMin1withCuda);
	validate(c[0], a, arraySize);
	c = (int*)malloc(blockSize * sizeof(int));
	
	cudaStatus = reductionMinWithCuda(c, a, arraySize, threadSize, blockSize, reductionMin2withCuda);
	validate(c[0], a, arraySize);
	c = (int*)malloc(blockSize * sizeof(int));

	cudaStatus = reductionMinWithCuda(c, a, arraySize, threadSize, blockSize, reductionMin3withCuda);
	validate(c[0], a, arraySize);
	c = (int*)malloc(blockSize * sizeof(int));

	cudaStatus = reductionMinWithCuda(c, a, arraySize, threadSize, blockSize2, reductionMin4withCuda);
	validate(c[0], a, arraySize);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t reductionMinWithCuda(int* c, const int* a, unsigned int size, int threadSize, int blockSize, void(*kernel) (dim3, dim3, int, int*, int*, int))
{
	hipDeviceProp_t	devProp;
	hipGetDeviceProperties(&devProp, 0);
	int* dev_a = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	checkError(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", dev_a, dev_c);

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_c);

	cudaStatus = hipMalloc((void**)&dev_c, blockSize * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_c, dev_c);

	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_c);

	// Launch a kernel on the GPU with one thread for each element.
	hipEvent_t start, stop;		//описываем переменные типа  hipEvent_t 
	float       gpuTime = 0.0f;
	// создаем события начала и окончания выполнения ядра 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//привязываем событие start  к данному месту 
	hipEventRecord(start, 0);

	dim3 threads(threadSize, 1, 1);
	dim3 blocks(blockSize, 1, 1);
	kernel(blocks, threads, threadSize, dev_a, dev_c, size);
	//reductionMin1 << <blocks, threads, threadSize * sizeof(int) >> > (dev_a, dev_c, size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	checkError(cudaStatus, "reductionMin4 launch failed: %s\n", dev_a, dev_c);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	checkError(cudaStatus, "hipDeviceSynchronize returned error code %d after launching reductionMin4!\n", dev_a, dev_c);

	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	// запрашиваем время между событиями 
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("time spent executing by the GPU: %.5f ms\n", gpuTime);
	// уничтожаем созданные события 
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, blockSize * sizeof(int), hipMemcpyDeviceToHost);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_c);

	for (size_t i = 1; i < blockSize; i++)
	{
		if (c[i] < c[0]) {
			c[0] = c[i];
		}
	}

	return cudaStatus;
}


void checkError(hipError_t cudaStatus, char* msg, int* dev_a, int* dev_c)
{
	if (cudaStatus == hipSuccess)
		return;

	printf(msg);
	hipFree(dev_a);
	hipFree(dev_c);
}

void printArray(int* a, int size)
{
	for (int i = 0; i < size; i++)
		printf("%d ", a[i]);
}

void printMatrix(int* matrix, int size)
{
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
			printf("%d ", matrix[i * size + j]);
		printf("\n");
	}
	printf("\n");
}

void validate(int result, int* array, int arraySize) {
	int v = array[0];
	for (size_t i = 0; i < arraySize; i++)
	{
		if (array[i] < v) {
			v = array[i];
		}
	}
	if (result != v) {
		printf("\nWrong answer. Real answear %d\n", v);
	}
	else {
		printf("\nAnswer success\n");
	}
}