#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

void printArray(int* a, int size);

void checkError(hipError_t cudaStatus, char* msg, int* dev_a, int* dev_c);

void validate(int* result, int* array, int arraySize);

hipError_t histogramWithCuda(int *c, const int *a, unsigned int size, int threadSize);

__global__ void histogramSmemAtomics(const int* in, int size, int NUM_BINS, int* out)
{
    // pixel coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // step in block
    int nx = blockDim.x * gridDim.x;

    // linear thread index within block
    int t = threadIdx.x;

    // total threads in block
    int nt = blockDim.x;

    // block index
    int bx = blockIdx.x;

    // initialize temporary accumulation array in shared memory
    extern __shared__ unsigned int smem[];
    for (int i = t; i < NUM_BINS; i += nt) {
        smem[i] = 0;
    }

    __syncthreads();
    // updates our block's partial histogram in shared memory
    for (int col = x; col < size; col += nx)
    {
        unsigned int r = (unsigned int)(in[col]);
        //printf("%d: %d\n", col, in[col]);
        atomicAdd(&smem[r], 1);
    }
    __syncthreads();

    // write partial histogram into the global memory
    out += bx * NUM_BINS;
    for (int i = t; i < NUM_BINS; i += nt) {
        out[i] = smem[i];
    }
}

__global__ void histogramFinalAccum(const int* in, int NUM_BINS, int NUM_PARTS, int* out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < NUM_BINS) {
        unsigned int total = 0;
        for (int j = 0; j < NUM_PARTS; j++) {
            total += in[i + NUM_BINS * j];
        }
        out[i] = total;
    }
}

int main()
{
    printf("Enter array size: ");
    int arraySize;
    scanf("%d", &arraySize);
    int threadSize = 256;
    int* a = (int*)malloc(arraySize * sizeof(int));
    int* c = (int*)malloc(256 * sizeof(int));
    for (size_t i = 0; i < arraySize; i++)
    {
        a[i] = rand() % 256;
    }

    // Add vectors in parallel.
    hipError_t cudaStatus = histogramWithCuda(c, a, arraySize, threadSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "histogramWithCuda failed!");
        return 1;
    }

    /*for (size_t i = 0; i < 256; i++)
    {
        printf("%d: %d\n", i, c[i]);
    }*/
    validate(c, a, arraySize);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t histogramWithCuda(int* c, const int* a, unsigned int size, int threadSize)
{
    int NUM_BINS = 256;
    int PART_SIZE = size < 300000 ? 3000 : 10000;
    int NUM_PARTS = ceil(double(size) / PART_SIZE);
    int *input = 0;
    int *dev_c = 0;
    int* result = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    checkError(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", input, dev_c);

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, 256 * NUM_PARTS * sizeof(int));
    checkError(cudaStatus, "hipMalloc failed!", input, dev_c);

    cudaStatus = hipMalloc((void**)&result, 256 * sizeof(int));
    checkError(cudaStatus, "hipMalloc failed!", input, dev_c);

    cudaStatus = hipMalloc((void**)&input, size * sizeof(int));
    checkError(cudaStatus, "hipMalloc failed!", input, dev_c);

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(input, a, size * sizeof(int), hipMemcpyHostToDevice);
    checkError(cudaStatus, "hipMemcpy failed!", input, dev_c);
    
    hipEvent_t start, stop;
    float       gpuTime = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    histogramSmemAtomics <<<NUM_PARTS, threadSize, NUM_BINS * sizeof(int) >> >(input, size, NUM_BINS, dev_c);
    cudaStatus = hipGetLastError();
    checkError(cudaStatus, "histogramSmemAtomics launch failed: %s\n", input, dev_c);

    histogramFinalAccum << <2, NUM_BINS >> > (dev_c, NUM_BINS, NUM_PARTS, result);
    cudaStatus = hipGetLastError();
    checkError(cudaStatus, "histogramFinalAccum launch failed: %s\n", input, dev_c);

    cudaStatus = hipDeviceSynchronize();
    checkError(cudaStatus, "hipDeviceSynchronize returned error code %d after launching multiplyKernel!\n", input, dev_c);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("time spent executing by the GPU: %.5f ms\n", gpuTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, result, 256 * sizeof(int), hipMemcpyDeviceToHost);
    checkError(cudaStatus, "hipMemcpy failed!", input, dev_c);

    
    return cudaStatus;
}


void checkError(hipError_t cudaStatus, char* msg, int* dev_a, int* dev_c)
{
    if (cudaStatus == hipSuccess)
        return;

    printf("cudaStatus: %d\n", cudaStatus);
    printf(msg);
    hipFree(dev_a);
    hipFree(dev_c);
}

void printArray(int* a, int size)
{
    for (int i = 0; i < size; i++)
        printf("%d ", a[i]);
}

void validate(int* result, int* array, int arraySize) {
    int* test = new int[256];
    for (int i = 0; i < 256; i++)
        test[i] = 0;
    for (int i = 0; i < arraySize; i++)
        test[array[i]]++;

    for (int i = 0; i < 256; i++)
    {
        if (result[i] != test[i]) {
            printf("\nWrong answer (%d: %d != %d)", i, test[i], result[i]);
        }
    }

    printf("\nAnswer success");
}