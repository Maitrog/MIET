#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <crt/math_functions.hpp>


void printArray(int* a, int size);

void checkError(hipError_t cudaStatus, char* msg, int* dev_a, int* dev_c);

void validate(int* result, int* array, int arraySize);

void prefixSumKernel(int* inData, int* outData, int& arraySize, int& blockSize, int sharedSize, void(*prefixSum) (int* inData, int* outData, int* sums, int arraySize));

void prefixSumKernel1(int* inData, int* outData, int& arraySize, int& blockSize);

void prefixSumKernel2(int* inData, int* outData, int& arraySize, int& blockSize);

hipError_t prefixSumWithCuda(int* c, const int* a, int size, int threadsCount, void (*prefixSumKernel) (int* inData, int* outData, int& arraySize, int& blockSize));

__global__ void prefixSum1(int* inData, int* outData, int* sums, int arraySize)
{
	extern __shared__ int temp[];
	int id = 2 * blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int offset = 1;
	int BLOCK_SIZE = blockDim.x;
	int sharedSize = 2 * BLOCK_SIZE;
	
	temp[tid] = inData[id]; // load into shared memory
	temp[tid + BLOCK_SIZE] = inData[id + BLOCK_SIZE];

	for (int d = sharedSize >> 1; d > 0; d >>= 1) {
		__syncthreads();
		if (tid < d)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset <<= 1;
	}
	if (tid == 0) {
		sums[blockIdx.x] = temp[sharedSize - 1];
		temp[sharedSize - 1] = 0; // clear the last element 
	}
	for (int d = 1; d < sharedSize; d <<= 1)
	{
		offset >>= 1;
		__syncthreads();
		if (tid < d)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	outData[2 * blockDim.x * blockIdx.x + 2 * tid] = temp[2 * tid]; // write results 
	outData[2 * blockDim.x * blockIdx.x + 2 * tid + 1] = temp[2 * tid + 1];
}

__global__ void prefixSum2(int* inData, int* outData, int* sums, int arraySize)
{
	extern __shared__ int temp[];
	int BLOCK_SIZE = blockDim.x * blockDim.y * blockDim.z;
	int tid = threadIdx.x;
	int offset = 1;
	int ai = tid; // 0 -> 255
	int bi = tid + (arraySize / 2); // 256 -> 511
	int offsA = (ai >> 4); // from 0 to 15
	int offsB = (bi >> 4); // from 16 to 31
	temp[ai + offsA] = inData[ai + 2 * BLOCK_SIZE * blockIdx.x];
	temp[bi + offsB] = inData[bi + 2 * BLOCK_SIZE * blockIdx.x];
	for (int d = arraySize >> 1; d > 0; d >>= 1, offset <<= 1)
	{
		__syncthreads();
		if (tid < d)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;
			ai += ai >> 4;
			bi += bi >> 4;
			temp[bi] += temp[ai];
		}
	}
	if (tid == 0)
	{
		int i = arraySize - 1 + ((arraySize - 1) >> 4);  // для scan больших массивов
		sums[blockIdx.x] = temp[i];             // для scan больших массивов
		temp[i] = 0; // clear the last element 
	}
	for (int d = 1; d < arraySize; d <<= 1)
	{
		offset >>= 1;
		__syncthreads();
		if (tid < d)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;
			int t;
			ai += (ai >> 4);
			bi += (bi >> 4);
			t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	outData[ai + 2 * BLOCK_SIZE * blockIdx.x] = temp[ai + offsA];
	outData[bi + 2 * BLOCK_SIZE * blockIdx.x] = temp[bi + offsB];
}

__global__ void finalPrefixSum(int* data, int* sums, int arraySize, int blockSize)
{
	int index = threadIdx.x + blockIdx.x * 2 * blockSize;
	if (index <= arraySize) {
		int v = data[index] + sums[blockIdx.x];
		data[index] = v;
	}
}


int main()
{
	printf("Enter array size: ");
	int arraySize;
	scanf("%d", &arraySize);
	int threadsCount = 64; // больше ставить не рекомендую, перестает работать стабильно работать, хз почему
	int* a = (int*)malloc(arraySize * sizeof(int));
	int* c = (int*)malloc(arraySize * sizeof(int));

	for (size_t i = 0; i < arraySize; i++)
	{
		a[i] = rand() * (rand() % 2 == 0 ? -1 : 1);
	}

	hipError_t cudaStatus = prefixSumWithCuda(c, a, arraySize, threadsCount, prefixSumKernel1);
	validate(c, a, arraySize);
	c = (int*)malloc(arraySize * sizeof(int));

	cudaStatus = prefixSumWithCuda(c, a, arraySize, threadsCount, prefixSumKernel1);
	validate(c, a, arraySize);
	c = (int*)malloc(arraySize * sizeof(int));

	cudaStatus = prefixSumWithCuda(c, a, arraySize, threadsCount, prefixSumKernel2);
	validate(c, a, arraySize);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "prefixSumWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

hipError_t prefixSumWithCuda(int* c, const int* a, int size, int threadsCount, void (*prefixSumKernel) (int* inData, int* outData, int& arraySize, int& blockSize))
{
	hipDeviceProp_t	devProp;
	hipGetDeviceProperties(&devProp, 0);
	int* dev_a = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	checkError(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", dev_a, dev_c);

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_c);

	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_c);

	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_c);

	hipEvent_t start, stop; //описываем переменные типа  hipEvent_t 
	float gpuTime = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// вызов основной функции вычисления префиксных сумм
	prefixSumKernel(dev_a, dev_c, size, threadsCount);

	cudaStatus = hipGetLastError();
	checkError(cudaStatus, "prefixSum2Kernel launch failed: %s\n", dev_a, dev_c);

	cudaStatus = hipDeviceSynchronize();
	checkError(cudaStatus, "hipDeviceSynchronize returned error code %d after launching prefixSum2Kernel!\n", dev_a, dev_c);

	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("time spent executing by the GPU: %.5f ms\n", gpuTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_c);

	return cudaStatus;
}

void prefixSumKernel(int* inData, int* outData, int& arraySize, int& blockSize, int sharedSize, void(*prefixSum) (int* inData, int* outData, int* sums, int arraySize))
{
	int numBlocks = ceil((double)arraySize / (2 * blockSize));
	int* sums; // суммы элементов для каждого блока, посчитанные для массива inData
	int* resultSums; // суммы элементов для каждого блока, посчитанные для массива sums (рекурсия)

	if (numBlocks < 1) {
		numBlocks = 1;
	}

	hipMalloc((void**)&sums, numBlocks * sizeof(int));
	hipMalloc((void**)&resultSums, numBlocks * sizeof(int));

	// вычисляем суммы для inData
	dim3 threads(blockSize, 1, 1), blocks(numBlocks, 1, 1);
	prefixSum<< <blocks, threads, sharedSize * sizeof(int) >> > (inData, outData, sums, 2 * blockSize);

	int* testRes = (int*)malloc(numBlocks * sizeof(int));
	hipMemcpy(testRes, sums, numBlocks * sizeof(int), hipMemcpyDeviceToHost);

	// вычисляем суммы для sums
	// тут рекурсия
	if (arraySize >= 2 * blockSize)
		prefixSumKernel(sums, resultSums, numBlocks, blockSize, sharedSize, prefixSum);
	else
		hipMemcpy(resultSums, sums, numBlocks * sizeof(int), hipMemcpyDeviceToDevice);

	hipError_t cudaStatus = hipGetLastError();
	// корректируем результат 
	threads = dim3(2 * blockSize, 1, 1);
	if (numBlocks <= 1) {
		blocks = dim3(1, 1, 1);
	}
	else {
		blocks = dim3(numBlocks - 1, 1, 1);
	}

	// передаем в функцию адреса со смещением
	// пропускаем первый блок
	finalPrefixSum << <blocks, threads >> > (outData + 2 * blockSize, resultSums + 1, arraySize, blockSize);
	cudaStatus = hipGetLastError();
	hipFree(resultSums);
	cudaStatus = hipGetLastError();
	hipFree(sums);
	cudaStatus = hipGetLastError();
}

void prefixSumKernel1(int* inData, int* outData, int& arraySize, int& blockSize) {
	int sharedSize = 2 * blockSize;
	prefixSumKernel(inData, outData, arraySize, blockSize, sharedSize, prefixSum1);
}

void prefixSumKernel2(int* inData, int* outData, int& arraySize, int& blockSize) {
	int sharedSize = 2 * blockSize + ((2 * blockSize) >> 4);
	prefixSumKernel(inData, outData, arraySize, blockSize, sharedSize, prefixSum2);

}

void checkError(hipError_t cudaStatus, char* msg, int* dev_a, int* dev_c)
{
	if (cudaStatus == hipSuccess)
		return;

	printf(msg);
	hipFree(dev_a);
	hipFree(dev_c);
}

void printArray(int* a, int size)
{
	for (int i = 0; i < size; i++)
		printf("%d ", a[i]);
}

void printMatrix(int* matrix, int size)
{
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
			printf("%d ", matrix[i * size + j]);
		printf("\n");
	}
	printf("\n");
}

void validate(int* result, int* array, int arraySize) {
	int v = 0;
	for (size_t i = 0; i < arraySize - 1; i++)
	{
		v += array[i];
		if (result[i+1] != v) {
			printf("\nWrong answer on step %d (%d != %d).", i+1, v, result[i+1]);
			//return;
			v = result[i + 1];
		}
	}
	printf("Answer success\n");
}