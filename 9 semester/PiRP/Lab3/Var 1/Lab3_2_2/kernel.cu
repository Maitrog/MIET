#include "hip/hip_runtime.h"
﻿/*
 * Parallel bitonic sort using CUDA.
 * Compile with
 * nvcc -arch=sm_11 bitonic_sort.cu
 * Based on http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
 * License: BSD 3
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

 /* Every thread gets exactly one value in the unsorted array. */
#define THREADS 256 // 2^9

void validate(float* result, int arraySize) {
    int v = 0;
    for (size_t i = 0; i < arraySize - 1; i++)
    {
        if (result[i] > result[i + 1]) {
            printf("Wrong answer\n");
            return;
        }
    }
    printf("\nAnswer success");
}

float random_float()
{
    return (float)rand() / (float)RAND_MAX;
}

void array_print(float* arr, int length)
{
    int i;
    for (i = 0; i < length; ++i) {
        printf("%1.3f ", arr[i]);
    }
    printf("\n");
}

void array_fill(float* arr, int length)
{
    //srand(time(NULL));
    int i;
    for (i = 0; i < length; ++i) {
        arr[i] = random_float();
    }
}

float* resizeArray(float* values, int arraySize, int newArraySize)
{
    float* newValues = (float*)malloc(newArraySize * sizeof(float));
    int d = newArraySize - arraySize;
    for (size_t i = 0; i < newArraySize; i++)
    {
        if (i < arraySize)
        {
            newValues[i] = values[i];
        }
        else
        {
            newValues[i] = INT_MAX;
        }
    }

    return newValues;
}

__global__ void bitonic_sort_step(float* dev_values, int j, int k)
{
    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i ^ j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj) > i) {
        if ((i & k) == 0) {
            /* Sort ascending */
            if (dev_values[i] > dev_values[ixj]) {
                /* exchange(i,ixj); */
                float temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
        if ((i & k) != 0) {
            /* Sort descending */
            if (dev_values[i] < dev_values[ixj]) {
                /* exchange(i,ixj); */
                float temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
    }
}

/**
 * Inplace bitonic sort using CUDA.
 */
float* bitonic_sort(float* values, int arraySize)
{
    int stepsCount = ceil(log2(arraySize));
    int newArraySize = pow(2, stepsCount);
    values = resizeArray(values, arraySize, newArraySize);

    float* dev_values;
    size_t size = newArraySize * sizeof(float);

    hipMalloc((void**)&dev_values, size);
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

    int blocksCount = ceil(newArraySize / THREADS);
    dim3 blocks(blocksCount, 1);    /* Number of blocks   */
    dim3 threads(THREADS, 1);  /* Number of threads  */
    
    int j, k;
    /* Major step */
    for (k = 2; k <= newArraySize; k <<= 1) {
        /* Minor step */
        for (j = k >> 1; j > 0; j = j >> 1) {
            bitonic_sort_step << <blocks, threads >> > (dev_values, j, k);
        }
    }
    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    hipFree(dev_values);

    return values;
}

int main(void)
{
    printf("Enter array size: ");
    int arraySize;
    scanf("%d", &arraySize);

    float* values = (float*)malloc(arraySize * sizeof(float));
    array_fill(values, arraySize);
    //array_print(values, arraySize);

    values = bitonic_sort(values, arraySize);

    printf("\n");
    //array_print(values, arraySize);
    validate(values, arraySize);
}
