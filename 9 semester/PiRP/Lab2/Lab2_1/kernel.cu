#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

void checkError(hipError_t cudaStatus, char* msg, int* dev_a, int* dev_b, int* dev_c);

hipError_t multiplyWithCuda(int* c, const int* a, const int* b, unsigned int size);

void printArray(int* a, int size);

__global__ void multiplyKernel(int* c, const int* a, const int* b)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadIdx.x == 30) {
		i+=10;
	}
	c[i] = a[i] + b[i];

}

int main()
{
	// вариант 1
	printf("Variant 1\n");
	printf("Enter array size: ");
	int arraySize;
	scanf("%d", &arraySize);
	int* a = (int*)malloc(arraySize * sizeof(int));
	int* b = (int*)malloc(arraySize * sizeof(int));
	int* c = (int*)malloc(arraySize * sizeof(int));
	for (size_t i = 0; i < arraySize; i++)
	{
		a[i] = rand() % 100 + 1;
		b[i] = rand() % 100 + 1;
		c[i] = 0;
	}

	// Add vectors in parallel.
	hipError_t cudaStatus = multiplyWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}


	//printf("A: ");
	//printArray(a, arraySize);

	//printf("\nB: ");
	//printArray(b, arraySize);

	//printf("\nC: ");
	//int sum = 0;
	//for (int i = 0; i < arraySize; i++) {
	//	sum += c[i];
	//}
	//printf("%d", sum);


	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t multiplyWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
	hipDeviceProp_t	devProp;
	hipGetDeviceProperties(&devProp, 0);
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	int dev_size = size;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	checkError(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", dev_a, dev_b, dev_c);

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, dev_size * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	cudaStatus = hipMalloc((void**)&dev_a, dev_size * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	cudaStatus = hipMalloc((void**)&dev_b, dev_size * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, dev_size * sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	cudaStatus = hipMemcpy(dev_b, b, dev_size * sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	cudaStatus = hipMemcpy(dev_c, c, dev_size * sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	//for (int offset = 0; offset < 128; offset++) {
		// Launch a kernel on the GPU with one thread for each element.
		hipEvent_t start, stop;		//описываем переменные типа  hipEvent_t 
		float       gpuTime = 0.0f;
		// создаем события начала и окончания выполнения ядра 
		hipEventCreate(&start);
		hipEventCreate(&stop);
		//привязываем событие start  к данному месту 
		hipEventRecord(start, 0);

		dim3 threads;
		if (dev_size < devProp.maxThreadsPerBlock) {
			threads = dim3(dev_size, 1, 1);
		}
		else {
			threads = dim3(devProp.maxThreadsPerBlock, 1, 1);
		}
		dev_size = 32;
		threads = dim3(32, 1, 1);
		dim3 blocks(ceil(dev_size / threads.x), 1, 1);
		multiplyKernel << <blocks, threads >> > (dev_c, dev_a, dev_b);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		checkError(cudaStatus, "multiplyKernel launch failed: %s\n", dev_a, dev_b, dev_c);

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		checkError(cudaStatus, "hipDeviceSynchronize returned error code %d after launching multiplyKernel!\n", dev_a, dev_b, dev_c);

		hipEventRecord(stop, 0);

		hipEventSynchronize(stop);
		// запрашиваем время между событиями 
		hipEventElapsedTime(&gpuTime, start, stop);
		printf("time spent executing by the GPU: %.5f ms\n", gpuTime);
		// уничтожаем созданные события 
		hipEventDestroy(start);
		hipEventDestroy(stop);

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(c, dev_c, dev_size * sizeof(int), hipMemcpyDeviceToHost);
	//}
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	return cudaStatus;
}


void checkError(hipError_t cudaStatus, char* msg, int* dev_a, int* dev_b, int* dev_c)
{
	if (cudaStatus == hipSuccess)
		return;

	printf(msg);
	printf(hipGetErrorString(cudaStatus));
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}

void printArray(int* a, int size)
{
	for (int i = 0; i < size; i++)
		printf("%d ", a[i]);
}