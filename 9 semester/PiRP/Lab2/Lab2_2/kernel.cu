#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

void checkError(hipError_t cudaStatus, char* msg, int* dev_a, int* dev_b, int* dev_c);

hipError_t multiplyWithCuda(int* c, const int* a, const int* b, unsigned int size);

void printMatrix(int* matrix, int size);




__global__ void multiplyKernel(int* c, int* a, int* b, unsigned int size)
{
	// BLOCK_SIZE^2 должно давать макс число потоков в блоке
	const int BLOCK_SIZE = 16;

	// индекс для выходной матрицы
	int index = blockIdx.y * blockDim.x * blockDim.y * gridDim.x + blockIdx.x * blockDim.y + threadIdx.y * blockDim.x * gridDim.x + threadIdx.x;
	
	// адрес выходного элемента, если представлять массив c в виде матрицы
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	// смещение, если размерность матрицы не кратна BLOCK_SIZE
	index -= (blockDim.x * gridDim.x - size) * j;

	int sum = 0;
	__shared__ int a_buff[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ int b_buff[BLOCK_SIZE][BLOCK_SIZE];
	for (size_t k = 0, bx = 0, by = 0;  k < BLOCK_SIZE * BLOCK_SIZE; k+=BLOCK_SIZE, bx++, by++)
	{
		// индексы элементов в подматрицах, в конце вычитание - смещение аналогичное переменной index
		int a_index = blockIdx.y * blockDim.x * blockDim.y * gridDim.x + bx * blockDim.y + threadIdx.y * blockDim.x * gridDim.x + threadIdx.x - (blockDim.x * gridDim.x - size) * j;
		int b_index = by * blockDim.x * blockDim.y * gridDim.x + blockIdx.x * blockDim.y + threadIdx.y * blockDim.x * gridDim.x + threadIdx.x - (blockDim.x * gridDim.x - size) * (by * blockDim.y + threadIdx.y);
		
		// контроль выхода за пределы размеров матрицы (если больше необходимого заполняем 0, так как не влияет на конечный результат)
		if (a_index >= size * (j + 1)) {
			a_buff[threadIdx.y][threadIdx.x] = 0;
		}
		else {
			a_buff[threadIdx.y][threadIdx.x] = a[a_index];
		}

		if (b_index >= size * (by * blockDim.y + threadIdx.y + 1)) {
			b_buff[threadIdx.y][threadIdx.x] = 0;
		}
		else {
			b_buff[threadIdx.y][threadIdx.x] = b[b_index];
		}

		__syncthreads();
		for (int k = 0; k < BLOCK_SIZE; k++)
		{
			sum += a_buff[threadIdx.y][k] * b_buff[k][threadIdx.x];
		}
		__syncthreads();
	}

	// контроль выхода за пределы размеров матрицы
	if (i >= size || j >= size)
		return;
	c[index] = sum;
}


int main()
{
	// вариант 0
	printf("Variant 0\n");
	int arraySize;
	scanf("%d", &arraySize);
	int* a = (int*)malloc(arraySize * arraySize * sizeof(int));
	int* b = (int*)malloc(arraySize * arraySize * sizeof(int));
	int* c = (int*)malloc(arraySize * arraySize * sizeof(int));
	for (size_t i = 0; i < arraySize; i++)
	{
		for (size_t j = 0; j < arraySize; j++)
		{
			a[i * arraySize + j] = rand() % 100 + 1;
			b[i * arraySize + j] = rand() % 100 + 1;
			c[i * arraySize + j] = 0;
		}
	}


	// Add vectors in parallel.
	hipError_t cudaStatus = multiplyWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	/*printf("A:\n");
	printMatrix(a, arraySize);
	printf("B:\n");
	printMatrix(b, arraySize);
	printf("C:\n");
	printMatrix(c, arraySize);*/


	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t multiplyWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
	hipDeviceProp_t	devProp;
	hipGetDeviceProperties(&devProp, 0);
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	int dev_size = size * size;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	checkError(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", dev_a, dev_b, dev_c);

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, dev_size * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	cudaStatus = hipMalloc((void**)&dev_a, dev_size * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	cudaStatus = hipMalloc((void**)&dev_b, dev_size * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, dev_size * sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	cudaStatus = hipMemcpy(dev_b, b, dev_size * sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	cudaStatus = hipMemcpy(dev_c, c, dev_size * sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	// Launch a kernel on the GPU with one thread for each element.
	hipEvent_t start, stop;		//описываем переменные типа  hipEvent_t 
	float       gpuTime = 0.0f;
	// создаем события начала и окончания выполнения ядра 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//привязываем событие start  к данному месту 
	hipEventRecord(start, 0);

	double d;
	if (size < sqrt(devProp.maxThreadsPerBlock)) {
		d = size;
	}
	else {
		d = sqrt(devProp.maxThreadsPerBlock);
	}
	d = 16;
	dim3 threads(d, d, 1);
	dim3 blocks(ceil(size / d), ceil(size / d), 1);
	multiplyKernel << <blocks, threads >> > (dev_c, dev_a, dev_b, size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	checkError(cudaStatus, "multiplyKernel launch failed: %s\n", dev_a, dev_b, dev_c);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	checkError(cudaStatus, "hipDeviceSynchronize returned error code %d after launching multiplyKernel!\n", dev_a, dev_b, dev_c);

	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	// запрашиваем время между событиями 
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("time spent executing by the GPU: %.5f ms\n", gpuTime);
	// уничтожаем созданные события 
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, dev_size * sizeof(int), hipMemcpyDeviceToHost);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_b, dev_c);

	return cudaStatus;
}


void checkError(hipError_t cudaStatus, char* msg, int* dev_a, int* dev_b, int* dev_c)
{
	if (cudaStatus == hipSuccess)
		return;

	printf(msg);
	printf(hipGetErrorString(cudaStatus));
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}

void printMatrix(int* matrix, int size)
{
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
			printf("%d ", matrix[i * size + j]);
		printf("\n");
	}
	printf("\n");
}